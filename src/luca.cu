#include "luca.h"

void processFrame(cv::Mat &frame, cv::VideoWriter &writer) {
    int originalType = frame.type();
    int originalStep = frame.step;
    frame.convertTo(frame, CV_32FC3);
    Npp32f *d_frame, *d_result_1;
    size_t pitch;

    // Allocate memory on the GPU, and copy the frame on it
    hipMallocPitch(&d_frame, &pitch, 3 * frame.cols * sizeof(Npp32f), frame.rows);
    hipMemcpy2D(d_frame, pitch, frame.ptr(), frame.step, 3 * frame.cols * sizeof(Npp32f), frame.rows, hipMemcpyHostToDevice);

    // Allocate memory for the green channel and extract it
    Npp32f* d_greenChannel;
    size_t greenPitch;
    hipMallocPitch(&d_result_1, &greenPitch, frame.cols * sizeof(Npp32f), frame.rows);
    hipMallocPitch(&d_greenChannel, &greenPitch, frame.cols * sizeof(Npp32f), frame.rows);

    NppiSize oSizeROI;
    oSizeROI.width = frame.cols;
    oSizeROI.height = frame.rows;

    // Extract the green channel
    nppiCopy_32f_C3C1R(d_frame + 1, pitch, d_greenChannel, greenPitch, oSizeROI);

    // Edge detection using Scharr
    nppiFilterScharrHoriz_32f_C1R(d_greenChannel, greenPitch, d_result_1, greenPitch, oSizeROI);

    // Convert the single channel on device to a 3-channel grayscale image for OpenCV on the hot
    cv::Mat channelResult(frame.rows, frame.cols, CV_32FC1);
    cv::Mat grayscaleImage(frame.rows, frame.cols, originalType);

    hipMemcpy2D(channelResult.ptr(), channelResult.step, d_result_1, greenPitch, frame.cols * sizeof(Npp32f), frame.rows, hipMemcpyDeviceToHost);
    channelResult.convertTo(channelResult, CV_8UC1);
    cv::cvtColor(channelResult, grayscaleImage, cv::COLOR_GRAY2BGR);

    // Save the frame in hte video
    writer.write(grayscaleImage);

    // Deallocate
    hipFree(d_frame);
    hipFree(d_result_1);
}

__host__ int main(int argc, char** argv) {
    printf("Submission of Luca Venturi\n");

    if (argc != 3) {
        printf("This programs requires 2 parameters instead of %d: the source video and the output video.", argc);
        return 1;
    }

    cv::VideoCapture cap(argv[1]);
    if (!cap.isOpened()) {
        std::cerr << "Error: Could not open video." << std::endl;
        return -1;
    }

    int frame_width = static_cast<int>(cap.get(cv::CAP_PROP_FRAME_WIDTH));
    int frame_height = static_cast<int>(cap.get(cv::CAP_PROP_FRAME_HEIGHT));
    double fps = cap.get(cv::CAP_PROP_FPS);

    cv::VideoWriter writer(argv[2], cv::VideoWriter::fourcc('m', 'p', '4', 'v'), fps, cv::Size(frame_width, frame_height));
    if (!writer.isOpened()) {
        std::cerr << "Error: Could not open destination video." << std::endl;
        return -1;
    }

    cv::Mat frame;
    int numFrame = 0;
    while (true) {
        // Read a new frame
        if (!cap.read(frame)) {
            break; // Break the loop if there are no more frames
        }

        printf("Frame %d\n", numFrame);
        numFrame++;

        processFrame(frame, writer);
    }
}